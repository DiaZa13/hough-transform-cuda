#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : make
 ============================================================================
 */
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "common/pgm.h"

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;

//*****************************************************************
// The CPU function returns a pointer to the accummulator
void CPU_HoughTran (unsigned char *pic, int w, int h, int **acc)
{
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  *acc = new int[rBins * degreeBins];
  memset (*acc, 0, sizeof (int) * rBins * degreeBins);
  int xCent = w / 2;
  int yCent = h / 2;
  float rScale = 2 * rMax / rBins;

  for (int i = 0; i < w; i++)
    for (int j = 0; j < h; j++)
      {
        int idx = j * w + i;
        if (pic[idx] > 0)
          {
            int xCoord = i - xCent;
            int yCoord = yCent - j;
            float theta = 0;
            for (int tIdx = 0; tIdx < degreeBins; tIdx++)
              {
                float r = xCoord * cos (theta) + yCoord * sin (theta);
                int rIdx = (r + rMax) / rScale;
                (*acc)[rIdx * degreeBins + tIdx]++;
                theta += radInc;
              }
          }
      }
}

__global__ void GPU_HoughTran (unsigned char *pic, int w, int h, int *acc, float rMax, float rScale, float *d_Cos, float *d_Sin)
{
  int gloID = blockDim.x * blockIdx.x + threadIdx.x;
  if (gloID >= w * h) return;      

  int xCent = w / 2;
  int yCent = h / 2;

  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  if (pic[gloID] > 0)
    {
      for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
          float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
          int rIdx = (r + rMax) / rScale;
          atomicAdd (acc + (rIdx * degreeBins + tIdx), 1);
        }
    }
}


int main (int argc, char **argv)
{
  int i;

  PGMImage inImg (argv[1]);

  int *cpuht;
  int w = inImg.x_dim;
  int h = inImg.y_dim;

  float* d_Cos;
  float* d_Sin;

  hipMalloc ((void **) &d_Cos, sizeof (float) * degreeBins);
  hipMalloc ((void **) &d_Sin, sizeof (float) * degreeBins);

  // CPU calculation
  CPU_HoughTran(inImg.pixels, w, h, &cpuht);

  // pre-compute values to be stored
  float *pcCos = (float *) malloc (sizeof (float) * degreeBins);
  float *pcSin = (float *) malloc (sizeof (float) * degreeBins);
  float rad = 0;
  for (i = 0; i < degreeBins; i++)
  {
    pcCos[i] = cos (rad);
    pcSin[i] = sin (rad);
    rad += radInc;
  }

  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  float rScale = 2 * rMax / rBins;

  // copy values to device
  hipMemcpy(d_Cos, pcCos, sizeof (float) * degreeBins, hipMemcpyHostToDevice);
  hipMemcpy(d_Sin, pcSin, sizeof (float) * degreeBins, hipMemcpyHostToDevice);

  // setup and copy data from host to device
  unsigned char *d_in, *h_in;
  int *d_hough, *h_hough;

  h_in = inImg.pixels;
  h_hough = (int *) malloc (degreeBins * rBins * sizeof (int));

  hipMalloc ((void **) &d_in, sizeof (unsigned char) * w * h);
  hipMalloc ((void **) &d_hough, sizeof (int) * degreeBins * rBins);
  hipMemcpy (d_in, h_in, sizeof (unsigned char) * w * h, hipMemcpyHostToDevice);
  hipMemset (d_hough, 0, sizeof (int) * degreeBins * rBins);

  int blockNum = ceil (w * h / 256.0);
  
  // Create CUDA events for timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  // Record the start event
  hipEventRecord(start, NULL);

  GPU_HoughTran <<< blockNum, 256 >>> (d_in, w, h, d_hough, rMax, rScale, d_Cos, d_Sin);
  
  // Record the stop event
  hipEventRecord(stop, NULL);
  
  // Wait for the stop event to complete
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  
  printf("Kernel execution time: %f milliseconds\n", milliseconds);

  // get results from device
  hipMemcpy (h_hough, d_hough, sizeof (int) * degreeBins * rBins, hipMemcpyDeviceToHost);

  // compare CPU and GPU results
  for (i = 0; i < degreeBins * rBins; i++)
  {
    if (cpuht[i] != h_hough[i])
      printf ("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
  }
  printf("Done!\n");

  // Clean-up
  free(h_hough);
  free(cpuht);
  free(pcCos);
  free(pcSin);
  hipFree(d_in);
  hipFree(d_hough);
  hipFree(d_Cos);
  hipFree(d_Sin);

  return 0;
}
