#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : make
 ============================================================================
 */
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "common/pgm.h"
#include <vector>
#include <numeric>
#include <cmath>
#include <algorithm>
#include "opencv2/opencv.hpp"

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;

//*****************************************************************
// The CPU function returns a pointer to the accummulator
void CPU_HoughTran (unsigned char *pic, int w, int h, int **acc)
{
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  *acc = new int[rBins * degreeBins];
  memset (*acc, 0, sizeof (int) * rBins * degreeBins);
  int xCent = w / 2;
  int yCent = h / 2;
  float rScale = 2 * rMax / rBins;

  for (int i = 0; i < w; i++)
    for (int j = 0; j < h; j++)
      {
        int idx = j * w + i;
        if (pic[idx] > 0)
          {
            int xCoord = i - xCent;
            int yCoord = yCent - j;
            float theta = 0;
            for (int tIdx = 0; tIdx < degreeBins; tIdx++)
              {
                float r = xCoord * cos (theta) + yCoord * sin (theta);
                int rIdx = (r + rMax) / rScale;
                (*acc)[rIdx * degreeBins + tIdx]++;
                theta += radInc;
              }
          }
      }
}

__global__ void GPU_HoughTran (unsigned char *pic, int w, int h, int *acc, float rMax, float rScale, float *d_Cos, float *d_Sin)
{
  int gloID = blockDim.x * blockIdx.x + threadIdx.x;
  if (gloID >= w * h) return;      

  int xCent = w / 2;
  int yCent = h / 2;

  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  if (pic[gloID] > 0)
    {
      for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
          float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
          int rIdx = (r + rMax) / rScale;
          atomicAdd (acc + (rIdx * degreeBins + tIdx), 1);
        }
    }
}


int main (int argc, char **argv)
{
  int i;

  PGMImage inImg (argv[1]);

  int *cpuht;
  int w = inImg.x_dim;
  int h = inImg.y_dim;

  float* d_Cos;
  float* d_Sin;

  hipMalloc ((void **) &d_Cos, sizeof (float) * degreeBins);
  hipMalloc ((void **) &d_Sin, sizeof (float) * degreeBins);

  // CPU calculation
  CPU_HoughTran(inImg.pixels, w, h, &cpuht);

  // pre-compute values to be stored
  float *pcCos = (float *) malloc (sizeof (float) * degreeBins);
  float *pcSin = (float *) malloc (sizeof (float) * degreeBins);
  float rad = 0;
  for (i = 0; i < degreeBins; i++)
  {
    pcCos[i] = cos (rad);
    pcSin[i] = sin (rad);
    rad += radInc;
  }

  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  float rScale = 2 * rMax / rBins;

  // copy values to device
  hipMemcpy(d_Cos, pcCos, sizeof (float) * degreeBins, hipMemcpyHostToDevice);
  hipMemcpy(d_Sin, pcSin, sizeof (float) * degreeBins, hipMemcpyHostToDevice);

  // setup and copy data from host to device
  unsigned char *d_in, *h_in;
  int *d_hough, *h_hough;

  h_in = inImg.pixels;
  h_hough = (int *) malloc (degreeBins * rBins * sizeof (int));

  hipMalloc ((void **) &d_in, sizeof (unsigned char) * w * h);
  hipMalloc ((void **) &d_hough, sizeof (int) * degreeBins * rBins);
  hipMemcpy (d_in, h_in, sizeof (unsigned char) * w * h, hipMemcpyHostToDevice);
  hipMemset (d_hough, 0, sizeof (int) * degreeBins * rBins);

  int blockNum = ceil (w * h / 256.0);
  
  // Create CUDA events for timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  // Record the start event
  hipEventRecord(start, NULL);

  GPU_HoughTran <<< blockNum, 256 >>> (d_in, w, h, d_hough, rMax, rScale, d_Cos, d_Sin);
  
  // Record the stop event
  hipEventRecord(stop, NULL);
  
  // Wait for the stop event to complete
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  
  printf("Kernel execution time: %f milliseconds\n", milliseconds);

  // get results from device
  hipMemcpy (h_hough, d_hough, sizeof (int) * degreeBins * rBins, hipMemcpyDeviceToHost);

  // Calcula el promedio de los pesos
  double sum = std::accumulate(h_hough, h_hough + degreeBins * rBins, 0);
  double mean = sum / (degreeBins * rBins);

  // Calcula la desviación estándar de los pesos
  double sq_sum = std::inner_product(h_hough, h_hough + degreeBins * rBins, h_hough, 0.0);
  double stdev = std::sqrt(sq_sum / (degreeBins * rBins) - mean * mean);

  // El threshold será el promedio + 2 * desviación estándar
  double threshold = mean + 2 * stdev;

  // Dibuja las líneas cuyo peso es mayor que el threshold
  cv::Mat img = cv::imread(argv[1], cv::IMREAD_COLOR);

  for (i = 0; i < degreeBins * rBins; i++) {
    if (h_hough[i] > threshold) {
      float r = (i / degreeBins) * rScale - rMax;
      float theta = (i % degreeBins) * radInc;
      float a = std::cos(theta), b = std::sin(theta);
      int x0 = a*r + w/2, y0 = b*r + h/2;
      cv::Point pt1, pt2;
      pt1.x = cvRound(x0 + 1000*(-b));
      pt1.y = cvRound(y0 + 1000*(a));
      pt2.x = cvRound(x0 - 1000*(-b));
      pt2.y = cvRound(y0 - 1000*(a));
      cv::line(img, pt1, pt2, cv::Scalar(0,0,255), 1, cv::LINE_AA);
    }
  }

  cv::imwrite("outputs/output.png", img);

  // compare CPU and GPU results
  for (i = 0; i < degreeBins * rBins; i++)
  {
    if (cpuht[i] != h_hough[i])
      printf ("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
  }
  printf("Done!\n");

  // Clean-up
  free(h_hough);
  free(cpuht);
  free(pcCos);
  free(pcSin);
  hipFree(d_in);
  hipFree(d_hough);
  hipFree(d_Cos);
  hipFree(d_Sin);

  return 0;
}
